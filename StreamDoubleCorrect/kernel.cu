#include "hip/hip_runtime.h"
/**
* Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*/
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "util.h"


#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}


int main(void) {
	hipDeviceProp_t  prop;
	int whichDevice;
	CUDA_CHECK_RETURN(hipGetDevice(&whichDevice));
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, whichDevice));
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return 0;
	}

	hipEvent_t     start, stop;
	float           elapsedTime;

	hipStream_t    stream0, stream1;
	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	// start the timers
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	// initialize the streams
	CUDA_CHECK_RETURN(hipStreamCreate(&stream0));
	CUDA_CHECK_RETURN(hipStreamCreate(&stream1));

	// allocate the memory on the GPU
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a0,
		N * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_b0,
		N * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_c0,
		N * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a1,
		N * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_b1,
		N * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_c1,
		N * sizeof(int)));

	// allocate host locked memory, used to stream
	CUDA_CHECK_RETURN(hipHostAlloc((void**)&host_a,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipHostAlloc((void**)&host_b,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipHostAlloc((void**)&host_c,
		FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));

	for (int i = 0; i<FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	// now loop over full data, in bite-sized chunks
	for (int i = 0; i<FULL_DATA_SIZE; i += N * 2) {
		// enqueue copies


		// enqueue kernels in stream0 and stream1

		// enqueue copies of c from device to locked memory

	}
	CUDA_CHECK_RETURN(hipStreamSynchronize(stream0));
	CUDA_CHECK_RETURN(hipStreamSynchronize(stream1));

	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("Time taken:  %3.1f ms\n", elapsedTime);

	// cleanup the streams and memory
	CUDA_CHECK_RETURN(hipHostFree(host_a));
	CUDA_CHECK_RETURN(hipHostFree(host_b));
	CUDA_CHECK_RETURN(hipHostFree(host_c));
	CUDA_CHECK_RETURN(hipFree(dev_a0));
	CUDA_CHECK_RETURN(hipFree(dev_b0));
	CUDA_CHECK_RETURN(hipFree(dev_c0));
	CUDA_CHECK_RETURN(hipFree(dev_a1));
	CUDA_CHECK_RETURN(hipFree(dev_b1));
	CUDA_CHECK_RETURN(hipFree(dev_c1));
	CUDA_CHECK_RETURN(hipStreamDestroy(stream0));
	CUDA_CHECK_RETURN(hipStreamDestroy(stream1));

	return 0;
}


