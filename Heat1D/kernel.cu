#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include "book.h"
#include "cpu_anim.h"
#include "hip/hip_runtime.h"
#include "util.h"

#define BLOCK_SIZE 16

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f



// these exist on the GPU side
texture<float, hipTextureType1D, hipReadModeElementType>  texConstSrc;
texture<float, hipTextureType1D, hipReadModeElementType>  texIn;
texture<float, hipTextureType1D, hipReadModeElementType>  texOut;



// this kernel takes in a 2-d array of floats
// it updates the value-of-interest by a scaled value based
// on itself and its nearest neighbors
__global__ void blend_kernel(float *dst,
	bool dstOut) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0)   left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0)   top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	float   t, l, c, r, b;
	if (dstOut) {
		t = tex1Dfetch(texIn, top);
		l = tex1Dfetch(texIn, left);
		c = tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);

	}
	else {
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}


__global__ void copy_const_kernel(float *iptr) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstSrc, offset);

	// Preservar valores já computados
	if (c != 0)
		iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
	unsigned char   *output_bitmap;
	float           *dev_inSrc;
	float           *dev_outSrc;
	float           *dev_constSrc;
	CPUAnimBitmap  *bitmap;

	hipEvent_t     start, stop;
	float           totalTime;
	float           frames;
};

void anim_gpu(DataBlock *d, int ticks) {
	CUDA_CHECK_RETURN(hipEventRecord(d->start, 0));
	dim3    blocks(DIM / BLOCK_SIZE, DIM / BLOCK_SIZE);
	dim3    threads(BLOCK_SIZE, BLOCK_SIZE);
	CPUAnimBitmap  *bitmap = d->bitmap;

	// since tex is global and bound, we have to use a flag to
	// select which is in/out per iteration
	volatile bool dstOut = true;
	for (int i = 0; i<90; i++) {
		float   *in, *out;
		if (dstOut) {
			in = d->dev_inSrc;
			out = d->dev_outSrc;
		}
		else {
			out = d->dev_inSrc;
			in = d->dev_outSrc;
		}
		copy_const_kernel << <blocks, threads >> >(in);
		blend_kernel << <blocks, threads >> >(out, dstOut);
		dstOut = !dstOut;
	}
	float_to_color << <blocks, threads >> >(d->output_bitmap,
		d->dev_inSrc);

	CUDA_CHECK_RETURN(hipMemcpy(bitmap->get_ptr(),
		d->output_bitmap,
		bitmap->image_size(),
		hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipEventRecord(d->stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(d->stop));
	float   elapsedTime;
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime,
		d->start, d->stop));
	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average Time per frame:  %3.1f ms\n",
		d->totalTime / d->frames);
}

// clean up memory allocated on the GPU
void anim_exit(DataBlock *d) {
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);
	CUDA_CHECK_RETURN(hipFree(d->dev_inSrc));
	CUDA_CHECK_RETURN(hipFree(d->dev_outSrc));
	CUDA_CHECK_RETURN(hipFree(d->dev_constSrc));

	CUDA_CHECK_RETURN(hipEventDestroy(d->start));
	CUDA_CHECK_RETURN(hipEventDestroy(d->stop));
}


int main(void) {
	DataBlock   data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;
	CUDA_CHECK_RETURN(hipEventCreate(&data.start));
	CUDA_CHECK_RETURN(hipEventCreate(&data.stop));

	int imageSize = bitmap.image_size();

	CUDA_CHECK_RETURN(hipMalloc((void**)&data.output_bitmap,
		imageSize));

	// assume float == 4 chars in size (ie rgba)
	CUDA_CHECK_RETURN(hipMalloc((void**)&data.dev_inSrc,
		imageSize));
	CUDA_CHECK_RETURN(hipMalloc((void**)&data.dev_outSrc,
		imageSize));
	CUDA_CHECK_RETURN(hipMalloc((void**)&data.dev_constSrc,
		imageSize));

	CUDA_CHECK_RETURN(hipBindTexture(NULL, texConstSrc,
		data.dev_constSrc,
		imageSize));

	CUDA_CHECK_RETURN(hipBindTexture(NULL, texIn,
		data.dev_inSrc,
		imageSize));

	CUDA_CHECK_RETURN(hipBindTexture(NULL, texOut,
		data.dev_outSrc,
		imageSize));

	// intialize the constant data
	float *temp = (float*)malloc(imageSize);
	for (int i = 0; i<DIM*DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x>300) && (x<600) && (y>310) && (y<601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y<900; y++) {
		for (int x = 400; x<500; x++) {
			temp[x + y*DIM] = MIN_TEMP;
		}
	}
	CUDA_CHECK_RETURN(hipMemcpy(data.dev_constSrc, temp,
		imageSize,
		hipMemcpyHostToDevice));

	// initialize the input data
	for (int y = 800; y<DIM; y++) {
		for (int x = 0; x<200; x++) {
			temp[x + y*DIM] = MAX_TEMP;
		}
	}
	CUDA_CHECK_RETURN(hipMemcpy(data.dev_inSrc, temp,
		imageSize,
		hipMemcpyHostToDevice));
	free(temp);

	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu,
		(void(*)(void*))anim_exit);
}

